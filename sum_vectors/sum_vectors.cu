#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../common/book.h"
#include <ctime>

#define N 100000000

__global__ void vec_sum(float* a, float* b, float* c) {
	int bid = blockIdx.x;
	if (bid < N) {
		c[bid] = a[bid] + b[bid];
	}
}

int main(void) {

	float *a = (float *)malloc(N * sizeof(float));
	float *b = (float *)malloc(N * sizeof(float));
	float *c = (float *)malloc(N * sizeof(float));

	for (int i = 0; i < N; i++) {
		a[i] = -i * .5;
		b[i] = (i * i) * .25;
	}

	double start_cpu = clock();
	for (int i = 0; i < N; i++) {
		c[i] = a[i] +  b[i];
	}
	double end_cpu = clock();

	printf("cpu time is %f seconds\n", double(end_cpu - start_cpu) / CLOCKS_PER_SEC);

	double start_gpu = clock();
	float *dev_a;
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(float)));
	float *dev_b;
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(float)));
	float *dev_c;
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(float)));

	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

	//dim3 grid(N);
	vec_sum<<<N,1>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(float) * N, hipMemcpyDeviceToHost));

	double end_gpu = clock();
	printf("GPU time: %f seconds\n", (end_gpu - start_gpu) / CLOCKS_PER_SEC);
	printf("a[25] is %f, b[25] is %f, c[25] is %f\n",  a[25], b[25],  c[25]);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	free(a);
	free(b);
	free(c);

	return 0;
}